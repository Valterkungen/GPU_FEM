#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <cstdlib> 
#include <cmath>
#include <hip/hip_runtime.h>


__global__ void assembleStiffnessMatrix(int numElements, int numNodes, float h, const float* localMass, const float *localStiffness, const int *connectivity, float *globalMass, float *globalStiffness) {
    
    int elemIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (elemIdx < numElements) {

        int node1 = connectivity[2 * elemIdx];     // Node i
        int node2 = connectivity[2 * elemIdx + 1]; // Node j

        // Atomic add local stiffness to global stiffness
        atomicAdd(&globalStiffness[node1 * numNodes + node1], (1.f/h) * localStiffness[0]); 
        atomicAdd(&globalStiffness[node1 * numNodes + node2], (1.f/h) * localStiffness[1]); 
        atomicAdd(&globalStiffness[node2 * numNodes + node1], (1.f/h) * localStiffness[2]); 
        atomicAdd(&globalStiffness[node2 * numNodes + node2], (1.f/h) * localStiffness[3]); 

        // Atomic add local stiffness to global mass
        atomicAdd(&globalMass[node1 * numNodes + node1], (1.f/h) * localMass[0]); 
        atomicAdd(&globalMass[node1 * numNodes + node2], (1.f/h) * localMass[1]); 
        atomicAdd(&globalMass[node2 * numNodes + node1], (1.f/h) * localMass[2]); 
        atomicAdd(&globalMass[node2 * numNodes + node2], (1.f/h) * localMass[3]); 
    }
}

__global__ void setupConnectivity(int numElements, int *connectivity) {
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < numElements) {
        int node1 = idx;               // current node
        int node2 = (idx + 1) % numElements; // next node, wraps around

        // Each element connects two nodes
        connectivity[2 * idx] = node1;
        connectivity[2 * idx + 1] = node2;
    }
}

__global__ void setMatrices(const int matrix_size, float *matrix){
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < matrix_size){
        matrix[idx] = 0.f;
    }
}


int main(int argc, char **argv) {
    
    int numNodes = 5;
    int numElements = numNodes;	
    float h = (1.f / numNodes);

    float localMass[2][2] = {{1.0f/3, 1.0f/6}, {1.0f/6, 1.0f/3}};
    float localStiffness[2][2] = {{1, -1}, {-1, 1}};
    float *localMass_gpu, *localStiffness_gpu;
    float *mass_matrix, *stiff_matrix;
    float *mass_matrix_gpu, *stiff_matrix_gpu;
    int *connectivity_gpu;

    // Allocation size for grids and blocks
    dim3 block_shape(256);
    dim3 grid_shape((numElements + block_shape.x - 1) / block_shape.x);

    // Allocate memory on GPU
    hipMalloc(&mass_matrix_gpu, numNodes * numNodes * sizeof(float));
    hipMalloc(&stiff_matrix_gpu, numNodes * numNodes * sizeof(float));
    hipMalloc(&connectivity_gpu, 2 * numElements * sizeof(int));
    hipMalloc(&localMass_gpu, 4 * sizeof(float));
    hipMalloc(&localStiffness_gpu, 4 * sizeof(float));

    // Allocate memory on host
    mass_matrix = (float *)malloc(numNodes * numNodes * sizeof(float));
    stiff_matrix = (float *)malloc(numNodes * numNodes * sizeof(float));

    // Initialize matrices on GPU
    setMatrices<<<grid_shape, block_shape>>>(numNodes * numNodes, mass_matrix_gpu);
    setMatrices<<<grid_shape, block_shape>>>(numNodes * numNodes, stiff_matrix_gpu);
    setupConnectivity<<<grid_shape, block_shape>>>(numElements, connectivity_gpu);

    // Copy local matrices to GPU
    hipMemcpy(localMass_gpu, localMass, 4 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(localStiffness_gpu, localStiffness, 4 * sizeof(float), hipMemcpyHostToDevice);

    // Run kernel to assemble matrices
    assembleStiffnessMatrix<<<grid_shape, block_shape>>>(numElements, numNodes, h,  localMass_gpu, localStiffness_gpu, connectivity_gpu, mass_matrix_gpu, stiff_matrix_gpu);

    // Copy results back to host
    hipMemcpy(mass_matrix, mass_matrix_gpu, numNodes * numNodes * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(stiff_matrix, stiff_matrix_gpu, numNodes * numNodes * sizeof(float), hipMemcpyDeviceToHost);

    // Print stiffness matrix
    std::cout << "Stiffness Matrix:" << std::endl;
    for (int i = 0; i < numNodes; i++) {
        for (int j = 0; j < numNodes; j++) {
            printf("%.2f ", stiff_matrix[i * numNodes + j]);
        }
        printf("\n");
    }

    // Print mass matrix
    std::cout << "Mass Matrix:" << std::endl;
    for (int i = 0; i < numNodes; i++) {
	for (int j = 0; j < numNodes; j++) {
            printf("%.2f ", mass_matrix[i * numNodes + j]);
	}
	    printf("\n");
    }

    // Free all allocated memory
    hipFree(mass_matrix_gpu);
    hipFree(stiff_matrix_gpu);
    hipFree(connectivity_gpu);
    hipFree(localMass_gpu);
    hipFree(localStiffness_gpu);
    free(mass_matrix);
    free(stiff_matrix);
}
